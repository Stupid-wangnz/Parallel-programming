#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"
#include "files.h"

#define SOFTENING 1e-9f
#define BLOCK_SIZE 64
#define BLOCK_STEP 32
#define BLOCK_NUM 64
#define MOD(a,b) ((a) - (a) / (b) * (b))
/*
 * Each body contains x, y, and z coordinate positions,
 * as well as velocities in the x, y, and z directions.
 */

typedef struct { float x, y, z, vx, vy, vz; } Body;

/*
 * Calculate the gravitational impact of all bodies in the system
 * on all others.
 */

__global__ void bodyForce(Body *p, float dt, int n) {
        //每个线程算一个点的一部分
                int i=MOD(threadIdx.x+blockIdx.x*blockDim.x,n);
                                int nn=n/(BLOCK_STEP*BLOCK_SIZE);
                float Fx = 0.0f;
                                float Fy = 0.0f;
                                float Fz = 0.0f;
                                __shared__ float x_shared[BLOCK_SIZE];
                                __shared__ float y_shared[BLOCK_SIZE];
                                __shared__ float z_shared[BLOCK_SIZE];
                                float xi=p[i].x;
                                float yi=p[i].y;
                                float zi=p[i].z;//储存对应位置的数据
                                float dx,dy,dz,distSqr,invDist,invDist3;
                                int j=(blockIdx.x+blockIdx.x/BLOCK_NUM);

                #pragma unroll
                                while(nn--){
                                        //从当前块开始向前推进
                                        j=MOD(j,BLOCK_NUM);
                                    //获得当前块中的当前相对位置的数据，并写到shared中
                                        x_shared[threadIdx.x]=p[j*BLOCK_SIZE+threadIdx.x].x;
                                        y_shared[threadIdx.x]=p[j*BLOCK_SIZE+threadIdx.x].y;
                                        z_shared[threadIdx.x]=p[j*BLOCK_SIZE+threadIdx.x].z;
                                        //同步锁
                                        __syncthreads();
                                        for(int k=0;k<BLOCK_SIZE;k++){
                        dx = x_shared[k] - xi;
                        dy = y_shared[k] - yi;
                        dz = z_shared[k] - zi;
                        distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
                        invDist = rsqrtf(distSqr);
                        invDist3 = invDist * invDist * invDist;
                        Fx += dx * invDist3;
                                                Fy += dy * invDist3;
                                                Fz += dz * invDist3;
                                        }
                                        j+=BLOCK_STEP;
                                        //算完之后才能写入新数据
                                        __syncthreads();
                }
                                //原子加保证数据正确性
                                atomicAdd(&p[i].vx, dt * Fx);
                                atomicAdd(&p[i].vy, dt * Fy);
                                atomicAdd(&p[i].vz, dt * Fz);

}

__global__ void integrate_position(Body *p,float dt,int n){
        int i=threadIdx.x+blockIdx.x*blockDim.x;
                // integrate position
                p[i].x += p[i].vx*dt;
                p[i].y += p[i].vy*dt;
                p[i].z += p[i].vz*dt;
}


int main(const int argc, const char** argv) {
  
  int deviceId;
  int numberOfSMs;

  hipGetDevice(&deviceId);
  hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
  
    
  // The assessment will test against both 2<11 and 2<15.
  // Feel free to pass the command line argument 15 when you gernate ./nbody report files
  int nBodies = 2<<11;
  if (argc > 1) nBodies = 2<<atoi(argv[1]);

  // The assessment will pass hidden initialized values to check for correctness.
  // You should not make changes to these files, or else the assessment will not work.
  const char * initialized_values;
  const char * solution_values;

  if (nBodies == 2<<11) {
    initialized_values = "files/initialized_4096";
    solution_values = "files/solution_4096";
  } else { // nBodies == 2<<15
    initialized_values = "files/initialized_65536";
    solution_values = "files/solution_65536";
  }

  if (argc > 2) initialized_values = argv[2];
  if (argc > 3) solution_values = argv[3];

  const float dt = 0.01f; // Time step
  const int nIters = 10;  // Simulation iterations

 int bytes = nBodies * sizeof(Body);
  float *buf;
  hipHostMalloc((void **)&buf,bytes);

  hipMemPrefetchAsync(p, bytes, deviceId);
  
  read_values_from_file(initialized_values, buf, bytes);
  float *d_buf;
  hipMalloc((void **)&d_buf,bytes);
  Body *d_p=(Body *)d_buf;

  hipMemcpy(d_buf,buf,bytes,hipMemcpyHostToDevice);
  double totalTime = 0.0;

  /*
   * This simulation will run for 10 cycles of time, calculating gravitational
   * interaction amongst bodies, and adjusting their positions to reflect.
   */

  for (int iter = 0; iter < nIters; iter++) {
    StartTimer();

  /*
   * You will likely wish to refactor the work being done in `bodyForce`,
   * and potentially the work to integrate the positions.
   */
    hipStream_t stream;
    hipStreamCreate(&stream);
    
    
    bodyForce<<<BLOCK_NUM*BLOCK_STEP,BLOCK_SIZE,0,stream>>>(d_p, dt, nBodies);
    hipStreamDestroy(stream);
  /*
   * This position integration cannot occur until this round of `bodyForce` has completed.
   * Also, the next round of `bodyForce` cannot begin until the integration is complete.
   */
    integrate_position<<<nBodies/BLOCK_SIZE,BLOCK_SIZE>>>(d_p,dt,nBodies);
    hipDeviceSynchronize();
    if(iter==nIters-1)
       hipMemcpy(buf,d_buf,bytes,hipMemcpyDeviceToHost);

    const double tElapsed = GetTimer() / 1000.0;
    totalTime += tElapsed;
  }

  double avgTime = totalTime / (double)(nIters);
  float billionsOfOpsPerSecond = 1e-9 * nBodies * nBodies / avgTime;
  write_values_to_file(solution_values, buf, bytes);

  // You will likely enjoy watching this value grow as you accelerate the application,
  // but beware that a failure to correctly synchronize the device might result in
  // unrealistically high values.
  printf("%0.3f Billion Interactions / second", billionsOfOpsPerSecond);

  free(buf);
}
